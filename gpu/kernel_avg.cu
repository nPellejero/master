#include "hip/hip_runtime.h"
#include "helpers_cuda.h"
#include "kernel_avg.h"


static const unsigned int BLOCK_WIDTH = 8;
static const unsigned int BLOCK_HEIGHT = 8;

static const unsigned int GRID_WIDTH = 161;
//static const unsigned int GRID_HEIGHT = 1;


static __global__ void mult(float val,float* matrix, unsigned int height, unsigned int width)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    matrix[x] = matrix[x]*val;
	
}
 

void mult_gpu(float val,float* matrix, unsigned int height, unsigned int width)
{
   // configurar la grilla para el kernel
    dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
    dim3 grid(GRID_WIDTH);
    
    mult<<<grid, block>>>(val,matrix,height,width);
}
